#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

#include <helper_gl.h>
#include <GL/freeglut.h>

#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>

#include <helper_functions.h>
#include <hip/hip_runtime_api.h>         
#include <hip/hip_vector_types.h>
#include <thrust/sort.h>
#include <thrust/execution_policy.h>

#define MAX_EPSILON_ERROR 10.0f
#define THRESHOLD          0.30f
#define REFRESH_DELAY     10 //ms

////////////////////////////////////////////////////////////////////////////////
//! Parameters
////////////////////////////////////////////////////////////////////////////////
#define N 1024 * 4
#define GRID_SIZE 8
#define GRID_RANGE 1    
#define FISH_LENGTH 4.0f
#define FISH_WIDTH 2.0f

#define MAX_VELOCITY 2.0f
#define MIN_VELOCITY 1.2f
#define MAX_ACCELERATION 0.5f

#define SIGHT_ANGLE 3.1415f * 0.55f
#define SIGHT_RANGE 900.0f //squared
#define PROTECTED_RANGE 400.0f // squared

#define TURN_FACTOR 1.5f
#define COHESION_FACTOR 2.0f
#define ALIGNMENT_FACTOR 4.0f
#define SEPARATION_FACTOR 4.0f
////////////////////////////////////////////////////////////////////////////////
//! Parameters
////////////////////////////////////////////////////////////////////////////////

#define mapRange(a1,a2,b1,b2,s) (b1 + (s-a1)*(b2-b1)/(a2-a1))

// constants
const int window_width  = 1000;
const int window_height = 1000;

const int sea_width    = window_width / 2;
const int sea_height   = window_height / 2;

const float cell_width = window_width / GRID_SIZE;
const float cell_height = window_height / GRID_SIZE;

float *d_x, *d_y, *d_vx, *d_vy, *d_future_vx, *d_future_vy;
int *d_gridCell, *d_gridFish, *d_cellStart, *d_startCell, *d_endCell;

// vbo variables
GLuint vbo;
struct hipGraphicsResource *cuda_vbo_resource;
void *d_vbo_buffer = NULL;

int mouse_old_x, mouse_old_y;
int mouse_buttons = 0;
float rotate_x = 0.0, rotate_y = 0.0;
float translate_z = -1.0;

int fpsCount = 0;        // FPS count for averaging
int fpsLimit = 1;        // FPS limit for sampling
int g_Index = 0;
float avgFPS = 0.0f;
unsigned int frameCount = 0;

StopWatchInterface *timer = NULL;

// declarations
bool initGL(int *argc, char **argv);
void initCUDA();
void createVBO(GLuint *vbo, struct hipGraphicsResource **vbo_res,
               unsigned int vbo_res_flags);
void deleteVBO(GLuint *vbo, struct hipGraphicsResource *vbo_res);

void cleanup();
void keyboard(unsigned char key, int x, int y);
void mouse(int button, int state, int x, int y);
void motion(int x, int y);

void computeFPS();
void timerEvent(int value);
void display();
void runCuda(struct hipGraphicsResource **vbo_resource);


__global__ void setUnsortedGrid(float* x, float* y, int* gridCell, int* gridFish)
{
    unsigned int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int column = (x[tid] + sea_width) / cell_width;
    int row = (y[tid] + sea_height) / cell_height;
    gridCell[tid] = max(min(row * GRID_SIZE + column, GRID_SIZE * GRID_SIZE - 1), 0);
    gridFish[tid] = tid;

}

__global__ void prepareCellStart(int* gridCell, int* cellStart)
{
    unsigned int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if(tid == 0)
        cellStart[gridCell[0]] = 0;
    else if(gridCell[tid] != gridCell[tid - 1])
        cellStart[gridCell[tid]] = tid;
}


__global__ void kernel_normalize_velocity(float *vx, float *vy, float *fvx, float *fvy)
{
    unsigned int tid = threadIdx.x + blockIdx.x * blockDim.x;
    
    float tvx = fvx[tid];
    float tvy = fvy[tid];
    
    float speed = sqrt(tvx * tvx + tvy * tvy);
    if(speed > MAX_VELOCITY)
    {
        tvx = MAX_VELOCITY * tvx / speed;
        tvy = MAX_VELOCITY * tvy / speed;
    }
    else if(speed < MIN_VELOCITY)
    {
        tvx = MIN_VELOCITY * tvx / speed;
        tvy = MIN_VELOCITY * tvy / speed;
    }
    vx[tid] = tvx;
    vy[tid] = tvy;
}

__global__ void prepareStartEndCell(int* cellStart, int* startCell, int* endCell)
{
    unsigned int tid = threadIdx.x + blockIdx.x * blockDim.x;

    if(tid >= GRID_SIZE * GRID_SIZE)
        return;

    int startPos = tid - GRID_RANGE;
    int endPos = tid + GRID_RANGE + 1;

    int tidRow = tid / GRID_SIZE;
    if(startPos / GRID_SIZE < tidRow)
        startPos = tidRow * GRID_SIZE;
    if(endPos / GRID_SIZE > tidRow)
        endPos = tidRow * GRID_SIZE + GRID_SIZE;

    if(startPos < 0)
        startPos = 0;
    if(endPos >= GRID_SIZE * GRID_SIZE)
        endPos = GRID_SIZE * GRID_SIZE;

    while(startPos < GRID_SIZE * GRID_SIZE && cellStart[startPos] == -1)
        startPos++;
    while(endPos < GRID_SIZE * GRID_SIZE && cellStart[endPos] == -1)
        endPos++;
    



    startCell[tid] = startPos == GRID_SIZE * GRID_SIZE ? N : cellStart[startPos];
    endCell[tid] = endPos == GRID_SIZE * GRID_SIZE ? N : cellStart[endPos];
}

__global__ void kernel_prepare_move(float *x, float *y, float *vx, float *vy, float *fvx, float *fvy,
        int* gridFish, int* startCell, int* endCell, int* gridCell)
{
    unsigned int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int cell = gridCell[tid];
    tid = gridFish[tid];

    // prepare accumulators
    float l_cohesionX = 0, l_cohesionY = 0;
    float l_alignementX = 0, l_alignementY = 0;
    float l_separationX = 0, l_separationY = 0;
    float l_count = 0;

    // prepare variables
    float tvx = vx[tid];
    float tvy = vy[tid];

    float tx = x[tid];
    float ty = y[tid];

    for(int i = -GRID_RANGE; i <= GRID_RANGE; i++)
    {
        int newCell = cell + i * GRID_SIZE;
        if(newCell >= 0 && newCell < GRID_SIZE * GRID_SIZE)
        {
            int startPos = startCell[newCell];
            int endPos = endCell[newCell];

            for(int i = startPos; i < endPos; i++)
            {
                int another = gridFish[i];
                float ax = x[another];
                float ay = y[another];

                float dx = tx - ax;
                float dy = ty - ay;

                float d = dx * dx + dy * dy;
                
                if(d < SIGHT_RANGE && d > 0 && acos((-dx * tvx + -dy * tvy) / sqrt(d * (tvx * tvx + tvy * tvy)) ) < SIGHT_ANGLE)
                {
                    l_cohesionX += ax;
                    l_cohesionY += ay;
                    l_alignementX += vx[another];
                    l_alignementY += vy[another];

                    float dsqrt = sqrt(d);

                    l_separationX += dx / dsqrt;
                    l_separationY += dy / dsqrt;

                    l_count += 1;

                }
            }
        }
    }
    if(l_count > 0)
    {

        float nvx = l_separationX * SEPARATION_FACTOR +
         (l_alignementX / l_count - tvx) * ALIGNMENT_FACTOR + 
         (l_cohesionX / l_count - tx) * COHESION_FACTOR;
        float nvy = l_separationY * SEPARATION_FACTOR + 
         (l_alignementY / l_count - tvy) * ALIGNMENT_FACTOR + 
         (l_cohesionY / l_count - ty) * COHESION_FACTOR;

        float d = sqrt(nvx * nvx + nvy * nvy);

        if(d > 0.001f)
        {
            fvx[tid] = tvx + MAX_ACCELERATION / d * nvx;
            fvy[tid] = tvy + MAX_ACCELERATION / d * nvy;
        }
    }
}

__global__ void kernel_move(float *x, float *y, float *vx, float *vy)
{
    unsigned int tid = threadIdx.x + blockIdx.x * blockDim.x;
    float nx = x[tid] + vx[tid];
    float ny = y[tid] + vy[tid];

    // repair X velocity
    if(nx < -sea_width)
        vx[tid] = vx[tid] + TURN_FACTOR;
    else if(nx > sea_width)
        vx[tid] = vx[tid] - TURN_FACTOR;
    // repair Y velocity
    if(ny < -sea_height)
        vy[tid] = vy[tid] + TURN_FACTOR;
    else if(ny > sea_height)
        vy[tid] = vy[tid] - TURN_FACTOR;
    x[tid] = nx;
    y[tid] = ny;
    
}


__global__ void kernel_display(float *pos, float *x, float *y, float *vx, float *vy)
{
    unsigned int tid = threadIdx.x + blockIdx.x * blockDim.x;
    float p1X = x[tid];
    float p1Y = y[tid];

    float tvx = vx[tid];
    float tvy = vy[tid];

    float coef = FISH_LENGTH / sqrt(tvx * tvx + tvy * tvy);
    float tempX = coef * tvx;
    float tempY = coef * tvy;

    float p2X = p1X - tempX - tempY * FISH_WIDTH / FISH_LENGTH;
    float p2Y = p1Y - tempY + tempX * FISH_WIDTH / FISH_LENGTH;
    float p3X = p1X - tempX + tempY * FISH_WIDTH / FISH_LENGTH;
    float p3Y = p1Y - tempY - tempX * FISH_WIDTH / FISH_LENGTH;

    pos[9 * tid] = p1X / window_width;
    pos[9 * tid + 1] = p1Y / window_height;
    pos[9 * tid + 2] = 0.0f;
    pos[9 * tid + 3] = p2X / window_width;
    pos[9 * tid + 4] = p2Y / window_height;
    pos[9 * tid + 5] = 0.0f;    
    pos[9 * tid + 6] = p3X / window_width;
    pos[9 * tid + 7] = p3Y / window_height;
    pos[9 * tid + 8] = 0.0f;

}


int main(int argc, char **argv)
{
    // Create the CUTIL timer
    sdkCreateTimer(&timer);

    initCUDA();
    initGL(&argc, argv);

    createVBO(&vbo, &cuda_vbo_resource, cudaGraphicsMapFlagsWriteDiscard);
    glutMainLoop();

    return 0;
}

void initCUDA()
{
    float* h_x = new float[N];
    float* h_y = new float[N];
    float* h_vx = new float[N];
    float* h_vy = new float[N];

    // srand(time(NULL));
    srand(0);

    for(int i = 0; i < N; ++i)
    {
        h_x[i] = mapRange(0, 100, -sea_width, sea_width, rand() % 100);
        h_y[i] = mapRange(0, 100, -sea_height, sea_height, rand() % 100);

        // std::cout << h_x[i] << " " << h_y[i] << std::endl;

        h_vx[i] = mapRange(0, 100, -MAX_VELOCITY, MAX_VELOCITY, rand() % 100);
        h_vy[i] = mapRange(0, 100, -MAX_VELOCITY, MAX_VELOCITY, rand() % 100);
        // printf("Line %d has x: %f y: %f vx: %f vy: %f\n", i, h_x[i], h_y[i], h_vx[i], h_vy[i]);
    }
    hipMalloc(&d_x, N * sizeof(float));
    hipMalloc(&d_y, N * sizeof(float));
    hipMalloc(&d_vx, N * sizeof(float));
    hipMalloc(&d_vy, N * sizeof(float));
    hipMalloc(&d_future_vx, N * sizeof(float));
    hipMalloc(&d_future_vy, N * sizeof(float));
    hipMalloc(&d_gridCell, N * sizeof(int));
    hipMalloc(&d_gridFish, N * sizeof(int));
    hipMalloc(&d_cellStart, GRID_SIZE * GRID_SIZE * sizeof(int));
    hipMalloc(&d_startCell, GRID_SIZE * GRID_SIZE * sizeof(int));
    hipMalloc(&d_endCell, GRID_SIZE * GRID_SIZE * sizeof(int));

    hipMemcpy(d_x, h_x, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_y, h_y, N * sizeof(float), hipMemcpyHostToDevice);    
    hipMemcpy(d_vx, h_vx, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_vy, h_vy, N * sizeof(float), hipMemcpyHostToDevice);

    free(h_x);
    free(h_y);
    free(h_vx);
    free(h_vy);
}

bool initGL(int *argc, char **argv)
{
    glutInit(argc, argv);
    glutInitDisplayMode(GLUT_RGBA | GLUT_DOUBLE);
    glutInitWindowSize(window_width, window_height);
    glutCreateWindow("Fish simulation");
    glutDisplayFunc(display);
    glutKeyboardFunc(keyboard);
    glutMotionFunc(motion);
    glutTimerFunc(REFRESH_DELAY, timerEvent,0);

    // initialize necessary OpenGL extensions
    if (! isGLVersionSupported(2,0))
    {
        fprintf(stderr, "ERROR: Support for necessary OpenGL extensions missing.");
        fflush(stderr);
        return false;
    }

    // default initialization
    glClearColor(0.0, 0.0, 0.0, 1.0);
    glDisable(GL_DEPTH_TEST);

    // viewport
    glViewport(0, 0, window_width, window_height);

    // projection
    glMatrixMode(GL_PROJECTION);
    glLoadIdentity();
    gluPerspective(60.0, (GLfloat)window_width / (GLfloat) window_height, 0.1, 10.0);

    SDK_CHECK_ERROR_GL();

    glutMouseFunc(mouse);
    glutCloseFunc(cleanup);


    return true;
}

void timerEvent(int value)
{
    if (glutGetWindow())
    {
        glutPostRedisplay();
        glutTimerFunc(REFRESH_DELAY, timerEvent,0);
    }
}

void cleanup()
{

    if (vbo)
    {
        deleteVBO(&vbo, cuda_vbo_resource);
    }
    hipFree(d_x);
    hipFree(d_y);
    hipFree(d_vx);
    hipFree(d_vy);
    hipFree(d_future_vx);
    hipFree(d_future_vy);
}

void createVBO(GLuint *vbo, struct hipGraphicsResource **vbo_res,
               unsigned int vbo_res_flags)
{
    assert(vbo);

    // create buffer object
    glGenBuffers(1, vbo);
    glBindBuffer(GL_ARRAY_BUFFER, *vbo);

    // initialize buffer object
    unsigned int size = 9 * N * sizeof(float);
    glBufferData(GL_ARRAY_BUFFER, size, 0, GL_DYNAMIC_DRAW);

    glBindBuffer(GL_ARRAY_BUFFER, 0);

    // register this buffer object with CUDA
    checkCudaErrors(hipGraphicsGLRegisterBuffer(vbo_res, *vbo, vbo_res_flags));

    SDK_CHECK_ERROR_GL();
}

void deleteVBO(GLuint *vbo, struct hipGraphicsResource *vbo_res)
{

    // unregister this buffer object with CUDA
    checkCudaErrors(hipGraphicsUnregisterResource(vbo_res));

    glBindBuffer(1, *vbo);
    glDeleteBuffers(1, vbo);

    *vbo = 0;
}

void display()
{
    sdkStartTimer(&timer);

    // run CUDA kernel to generate vertex positions
    runCuda(&cuda_vbo_resource);

    glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);

    // set view matrix
    glMatrixMode(GL_MODELVIEW);
    glLoadIdentity();
    glTranslatef(0.0, 0.0, translate_z);
    glRotatef(rotate_x, 1.0, 0.0, 0.0);
    glRotatef(rotate_y, 0.0, 1.0, 0.0);

    // render from the vbo
    glBindBuffer(GL_ARRAY_BUFFER, vbo);
    glVertexPointer(3, GL_FLOAT, 0, 0);

    glEnableClientState(GL_VERTEX_ARRAY);
    glColor3f(1.0, 0.0, 0.0);
    glDrawArrays(GL_TRIANGLES, 0, 9 * N);
    glDisableClientState(GL_VERTEX_ARRAY);

    glutSwapBuffers();
    sdkStopTimer(&timer);
    computeFPS();
}

void runCuda(struct hipGraphicsResource **vbo_resource)
{
    // map OpenGL buffer object for writing from CUDA
    float *dptr;
    checkCudaErrors(hipGraphicsMapResources(1, vbo_resource, 0));
    size_t num_bytes;
    checkCudaErrors(hipGraphicsResourceGetMappedPointer((void **)&dptr, &num_bytes,
                                                         *vbo_resource));
    //printf("CUDA mapped VBO: May access %ld bytes\n", num_bytes);


    // execute the kernel
    dim3 grid2D(N/1024, 1, 1);
    dim3 block2D(1024, 1, 1);
    dim3 gridGridSize(GRID_SIZE * GRID_SIZE / 1024 + 1, 1, 1);
    dim3 blockGridSize(1024, 1, 1);     
    // dim3 grid3D(N / 32, N / 32, 1);
    // dim3 block3D(32, 32, 1);
    // dim3 gridReduce(N * N / 1024, 1, 1);
    // dim3 blockReduce(1024, 1, 1);

    // size_t shm_size = 1024 * sizeof(float);
    // {
    //     float* debug = new float[N];
    //     hipMemcpy(debug, d_x, N * sizeof(float), hipMemcpyDeviceToHost);
    //     for(int i = 0; i < N; i++)
    //     {
    //         std::cout << i << ") " << debug[i] << std::endl;
    //     }
    //     exit(1);
    // }
    // Prepare helping grid
    setUnsortedGrid<<<grid2D, block2D>>>(d_x, d_y, d_gridCell, d_gridFish);
    thrust::sort_by_key(thrust::device, d_gridCell, d_gridCell + N, d_gridFish);
    hipMemset(d_cellStart, -1, GRID_SIZE * GRID_SIZE * sizeof(int));
    prepareCellStart<<<grid2D, block2D>>>(d_gridCell, d_cellStart);
    prepareStartEndCell<<<gridGridSize, blockGridSize>>>(d_cellStart, d_startCell, d_endCell);
    // {
    //     int* debugCellStart = new int[GRID_SIZE * GRID_SIZE];
    //     int* debugStart = new int[GRID_SIZE * GRID_SIZE];
    //     int* debugEnd = new int[GRID_SIZE * GRID_SIZE];
    //     hipMemcpy(debugCellStart, d_cellStart, GRID_SIZE * GRID_SIZE * sizeof(int), hipMemcpyDeviceToHost);
    //     hipMemcpy(debugStart, d_startCell, GRID_SIZE * GRID_SIZE * sizeof(int), hipMemcpyDeviceToHost);
    //     hipMemcpy(debugEnd, d_endCell, GRID_SIZE * GRID_SIZE * sizeof(int), hipMemcpyDeviceToHost);
    //     for(int i = 0; i < GRID_SIZE * GRID_SIZE; i++)
    //     {
    //         std::cout << i << ") fishID: " << debugCellStart[i] << " start: " << debugStart[i] << " end: " << debugEnd[i] << std::endl;
    //     }
    //     exit(1);
    // }
    kernel_prepare_move<<<grid2D, block2D>>>(d_x, d_y, d_vx, d_vy, d_future_vx, d_future_vy, d_gridFish, d_startCell, d_endCell, d_gridCell);
    // {
    //     float* debug = new float[N];
    //     hipMemcpy(debug, d_count, N * sizeof(float), hipMemcpyDeviceToHost);
    //     for(int i = 0; i < N; i++)
    //     {
    //         std::cout << i << ") " << debug[i] << std::endl;
    //     }
    //     exit(1);
    // }
    kernel_normalize_velocity<<<grid2D, block2D>>>(d_vx, d_vy, d_future_vx, d_future_vy);
    kernel_move<<<grid2D, block2D>>>(d_x, d_y, d_vx, d_vy);
    kernel_display<<<grid2D, block2D>>>(dptr, d_x, d_y, d_vx, d_vy);

    // unmap buffer object
    checkCudaErrors(hipGraphicsUnmapResources(1, vbo_resource, 0));
}

////////////////////////////////////////////////////////////////////////////////
//! Keyboard events handler
////////////////////////////////////////////////////////////////////////////////
void keyboard(unsigned char key, int /*x*/, int /*y*/)
{
    switch (key)
    {
        case (27) :
            #if defined(__APPLE__) || defined(MACOSX)
                exit(EXIT_SUCCESS);
            #else
                glutDestroyWindow(glutGetWindow());
                return;
            #endif
    }
}

////////////////////////////////////////////////////////////////////////////////
//! Mouse event handlers
////////////////////////////////////////////////////////////////////////////////
void mouse(int button, int state, int x, int y)
{
    if (state == GLUT_DOWN)
    {
        mouse_buttons |= 1<<button;
    }
    else if (state == GLUT_UP)
    {
        mouse_buttons = 0;
    }

    mouse_old_x = x;
    mouse_old_y = y;
}

void motion(int x, int y)
{
    float dx, dy;
    dx = (float)(x - mouse_old_x);
    dy = (float)(y - mouse_old_y);

    if (mouse_buttons & 1)
    {
        rotate_x += dy * 0.2f;
        rotate_y += dx * 0.2f;
    }
    else if (mouse_buttons & 4)
    {
        translate_z += dy * 0.01f;
    }

    mouse_old_x = x;
    mouse_old_y = y;
}

void computeFPS()
{
    frameCount++;
    fpsCount++;

    if (fpsCount == fpsLimit)
    {
        avgFPS = 1.f / (sdkGetAverageTimerValue(&timer) / 1000.f);
        fpsCount = 0;
        fpsLimit = (int)MAX(avgFPS, 1.f);

        sdkResetTimer(&timer);
    }

    char fps[256];
    sprintf(fps, "Fish simulation: %3.1f fps (Max 100Hz)", avgFPS);
    glutSetWindowTitle(fps);
}