#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

#include <helper_gl.h>
#include <GL/freeglut.h>

#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>

#include <helper_functions.h>
#include <hip/hip_runtime_api.h>         
#include <hip/hip_vector_types.h>

#define MAX_EPSILON_ERROR 10.0f
#define THRESHOLD          0.30f
#define REFRESH_DELAY     10 //ms

#define N 1024
#define FISH_LENGTH 8.0f
#define FISH_WIDTH 3.0f

#define MAX_VELOCITY 1.0f
#define MIN_VELOCITY 0.8f

#define MAX_ACCELERATION 0.2f

#define TURN_FACTOR 0.04f

#define SIGHT 10000.0f

#define mapRange(a1,a2,b1,b2,s) (b1 + (s-a1)*(b2-b1)/(a2-a1))

// constants
const int window_width  = 1024;
const int window_height = 960;

const int sea_width    = window_width / 2;
const int sea_height   = window_height / 2;

float *d_x, *d_y, *d_vx, *d_vy, *d_tempx, *d_tempy, *d_count;

// vbo variables
GLuint vbo;
struct hipGraphicsResource *cuda_vbo_resource;
void *d_vbo_buffer = NULL;

int mouse_old_x, mouse_old_y;
int mouse_buttons = 0;
float rotate_x = 0.0, rotate_y = 0.0;
float translate_z = -1.0;

int fpsCount = 0;        // FPS count for averaging
int fpsLimit = 1;        // FPS limit for sampling
int g_Index = 0;
float avgFPS = 0.0f;
unsigned int frameCount = 0;

StopWatchInterface *timer = NULL;

// declarations
bool initGL(int *argc, char **argv);
void initCUDA();
void createVBO(GLuint *vbo, struct hipGraphicsResource **vbo_res,
               unsigned int vbo_res_flags);
void deleteVBO(GLuint *vbo, struct hipGraphicsResource *vbo_res);

void cleanup();
void keyboard(unsigned char key, int x, int y);
void mouse(int button, int state, int x, int y);
void motion(int x, int y);

void computeFPS();
void timerEvent(int value);
void display();
void runCuda(struct hipGraphicsResource **vbo_resource);

__global__ void kernel_normalize_velocity(float *vx, float *vy)
{
    unsigned int tid = threadIdx.x + blockIdx.x * blockDim.x;
    
    float tvx = vx[tid];
    float tvy = vy[tid];
    
    float speed = sqrt(tvx * tvx + tvy * tvy);
    if(speed > MAX_VELOCITY)
    {
        vx[tid] = MAX_VELOCITY * tvx / speed;
        vy[tid] = MAX_VELOCITY * tvy / speed;
    }
    else if(speed < MIN_VELOCITY)
    {
        vx[tid] = MIN_VELOCITY * tvx / speed;
        vy[tid] = MIN_VELOCITY * tvy / speed;
    }
}

__global__ void kernel_update_velocity(float *vx, float *vy, float *correctionX, float *correctionY, float *count)
{
    unsigned int tid = threadIdx.x + blockIdx.x * blockDim.x;

    if(count[tid] != 0)
    {
        float tvx = vx[tid];
        float tvy = vy[tid];

        float nvx = correctionX[N * tid] / count[tid] - tvx;
        float nvy = correctionY[N * tid] / count[tid] - tvy;

        float d = sqrt(nvx * nvx + nvy * nvy);

        nvx = tvx + MAX_ACCELERATION / d * nvx;
        nvy = tvy + MAX_ACCELERATION / d * nvy;

        vx[tid] = nvx;
        vy[tid] = nvy;

        // d = sqrt(nvx * nvx + nvy * nvy);


        // vx[tid] = MAX_VELOCITY / d * nvx;
        // vy[tid] = MAX_VELOCITY / d * nvy;
    }
}

__global__ void kernel_reduce3D(float *data_in, float *data_out)
{
    extern __shared__ int sdata[];
    // each thread loads one element from global to shared mem
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;
    sdata[tid] = data_in[i];
    __syncthreads();
    // do reduction in shared mem
    for (unsigned int s=blockDim.x/2; s>0; s>>=1) {
        if (tid < s) {
            sdata[tid] += sdata[tid + s];
        }
        __syncthreads();
    }
    // write result for this block to global mem
    if (tid == 0) data_out[blockDim.x * blockIdx.x] = sdata[0];
}

__global__ void kernel_prepare_move(float *x, float *y, float *vx, float *vy, float *correctionX, float *correctionY, float *count)
{
    unsigned int tidx = threadIdx.x + blockIdx.x * blockDim.x; // Fish number tidx is ANOTHER fish
    unsigned int tidy = threadIdx.y + blockIdx.y * blockDim.y; // Fish number tidy is MY fish

    float dx = x[tidy] - x[tidx];
    float dy = y[tidy] - y[tidx];

    float d = dx * dx + dy * dy;

    if(d < SIGHT)
    {
        correctionX[tidy * N + tidx] = vx[tidx]; //vx[tidx] + dx + x[tidx];
        correctionY[tidy * N + tidx] = vy[tidx]; //vy[tidx] + dy + y[tidx];

        count[tidy * N + tidx] = 1;
    }
    else
    {
        correctionX[tidy * N + tidx] = 0;
        correctionY[tidy * N + tidx] = 0;

        count[tidy * N + tidx] = 0;
    }

}

__global__ void kernel_move(float *x, float *y, float *vx, float *vy)
{
    unsigned int tid = threadIdx.x + blockIdx.x * blockDim.x;
    float nx = x[tid] + vx[tid];
    float ny = y[tid] + vy[tid];
    // // repair X
    // if(nx < -sea_width)
    //     nx = sea_width;
    // else if(nx > sea_width)
    //     nx = -sea_width;
    // // repair Y
    // if(ny < -sea_height)
    //     ny = sea_height;
    // else if(ny > sea_height)
    //     ny = -sea_height;
    if(nx < -sea_width)
        vx[tid] = vx[tid] + TURN_FACTOR * (-sea_width - nx);
    else if(nx > sea_width)
        vx[tid] = vx[tid] - TURN_FACTOR * (nx - sea_height);
    // repair Y
    if(ny < -sea_height)
        vy[tid] = vy[tid] + TURN_FACTOR * (-sea_height - ny);
    else if(ny > sea_height)
        vy[tid] = vy[tid] - TURN_FACTOR * (ny - sea_height);
    x[tid] = nx;
    y[tid] = ny;
    
}

__global__ void kernel_display(float *pos, float *x, float *y, float *vx, float *vy)
{
    unsigned int tid = threadIdx.x + blockIdx.x * blockDim.x;
    float p1X = x[tid];
    float p1Y = y[tid];

    float tvx = vx[tid];
    float tvy = vy[tid];

    float coef = FISH_LENGTH / sqrt(tvx * tvx + tvy * tvy);
    float tempX = coef * tvx;
    float tempY = coef * tvy;

    float p2X = p1X - tempX - tempY * FISH_WIDTH / FISH_LENGTH;
    float p2Y = p1Y - tempY + tempX * FISH_WIDTH / FISH_LENGTH;
    float p3X = p1X - tempX + tempY * FISH_WIDTH / FISH_LENGTH;
    float p3Y = p1Y - tempY - tempX * FISH_WIDTH / FISH_LENGTH;

    pos[9 * tid] = p1X / window_width;
    pos[9 * tid + 1] = p1Y / window_height;
    pos[9 * tid + 2] = 0.0f;
    pos[9 * tid + 3] = p2X / window_width;
    pos[9 * tid + 4] = p2Y / window_height;
    pos[9 * tid + 5] = 0.0f;    
    pos[9 * tid + 6] = p3X / window_width;
    pos[9 * tid + 7] = p3Y / window_height;
    pos[9 * tid + 8] = 0.0f;

}

int main(int argc, char **argv)
{
    // Create the CUTIL timer
    sdkCreateTimer(&timer);

    initCUDA();
    initGL(&argc, argv);

    createVBO(&vbo, &cuda_vbo_resource, cudaGraphicsMapFlagsWriteDiscard);
    glutMainLoop();

    return 0;
}

void initCUDA()
{
    float* h_x = new float[N];
    float* h_y = new float[N];
    float* h_vx = new float[N];
    float* h_vy = new float[N];

    // srand(time(NULL));
    srand(0);

    for(int i = 0; i < N; ++i)
    {
        h_x[i] = (rand() % window_width) - sea_width;
        h_y[i] = (rand() % window_height) - sea_height;

        h_vx[i] = mapRange(0, 100, -MAX_VELOCITY, MAX_VELOCITY, rand() % 100);
        h_vy[i] = mapRange(0, 100, -MAX_VELOCITY, MAX_VELOCITY, rand() % 100);
        // printf("Line %d has x: %f y: %f vx: %f vy: %f\n", i, h_x[i], h_y[i], h_vx[i], h_vy[i]);
    }

    hipMalloc(&d_x, N * sizeof(float));
    hipMalloc(&d_y, N * sizeof(float));
    hipMalloc(&d_vx, N * sizeof(float));
    hipMalloc(&d_vy, N * sizeof(float));
    hipMalloc(&d_tempx, N * N * sizeof(float));
    hipMalloc(&d_tempy, N * N * sizeof(float));
    hipMalloc(&d_count, N * N * sizeof(float));

    hipMemcpy(d_x, h_x, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_y, h_y, N * sizeof(float), hipMemcpyHostToDevice);    
    hipMemcpy(d_vx, h_vx, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_vy, h_vy, N * sizeof(float), hipMemcpyHostToDevice);
}

bool initGL(int *argc, char **argv)
{
    glutInit(argc, argv);
    glutInitDisplayMode(GLUT_RGBA | GLUT_DOUBLE);
    glutInitWindowSize(window_width, window_height);
    glutCreateWindow("Fish simulation");
    glutDisplayFunc(display);
    glutKeyboardFunc(keyboard);
    glutMotionFunc(motion);
    glutTimerFunc(REFRESH_DELAY, timerEvent,0);

    // initialize necessary OpenGL extensions
    if (! isGLVersionSupported(2,0))
    {
        fprintf(stderr, "ERROR: Support for necessary OpenGL extensions missing.");
        fflush(stderr);
        return false;
    }

    // default initialization
    glClearColor(0.0, 0.0, 0.0, 1.0);
    glDisable(GL_DEPTH_TEST);

    // viewport
    glViewport(0, 0, window_width, window_height);

    // projection
    glMatrixMode(GL_PROJECTION);
    glLoadIdentity();
    gluPerspective(60.0, (GLfloat)window_width / (GLfloat) window_height, 0.1, 10.0);

    SDK_CHECK_ERROR_GL();

    glutMouseFunc(mouse);
    glutCloseFunc(cleanup);


    return true;
}

void timerEvent(int value)
{
    if (glutGetWindow())
    {
        glutPostRedisplay();
        glutTimerFunc(REFRESH_DELAY, timerEvent,0);
    }
}

void cleanup()
{

    if (vbo)
    {
        deleteVBO(&vbo, cuda_vbo_resource);
    }
    hipFree(d_x);
    hipFree(d_y);
    hipFree(d_vx);
    hipFree(d_vy);
    hipFree(d_tempx);
    hipFree(d_tempy);
    hipFree(d_count);
}

void createVBO(GLuint *vbo, struct hipGraphicsResource **vbo_res,
               unsigned int vbo_res_flags)
{
    assert(vbo);

    // create buffer object
    glGenBuffers(1, vbo);
    glBindBuffer(GL_ARRAY_BUFFER, *vbo);

    // initialize buffer object
    unsigned int size = 9 * N * sizeof(float);
    glBufferData(GL_ARRAY_BUFFER, size, 0, GL_DYNAMIC_DRAW);

    glBindBuffer(GL_ARRAY_BUFFER, 0);

    // register this buffer object with CUDA
    checkCudaErrors(hipGraphicsGLRegisterBuffer(vbo_res, *vbo, vbo_res_flags));

    SDK_CHECK_ERROR_GL();
}

void deleteVBO(GLuint *vbo, struct hipGraphicsResource *vbo_res)
{

    // unregister this buffer object with CUDA
    checkCudaErrors(hipGraphicsUnregisterResource(vbo_res));

    glBindBuffer(1, *vbo);
    glDeleteBuffers(1, vbo);

    *vbo = 0;
}

void display()
{
    sdkStartTimer(&timer);

    // run CUDA kernel to generate vertex positions
    runCuda(&cuda_vbo_resource);

    glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);

    // set view matrix
    glMatrixMode(GL_MODELVIEW);
    glLoadIdentity();
    glTranslatef(0.0, 0.0, translate_z);
    glRotatef(rotate_x, 1.0, 0.0, 0.0);
    glRotatef(rotate_y, 0.0, 1.0, 0.0);

    // render from the vbo
    glBindBuffer(GL_ARRAY_BUFFER, vbo);
    glVertexPointer(3, GL_FLOAT, 0, 0);

    glEnableClientState(GL_VERTEX_ARRAY);
    glColor3f(1.0, 0.0, 0.0);
    glDrawArrays(GL_TRIANGLES, 0, 9 * N);
    glDisableClientState(GL_VERTEX_ARRAY);

    glutSwapBuffers();
    sdkStopTimer(&timer);
    computeFPS();
}

void runCuda(struct hipGraphicsResource **vbo_resource)
{
    // map OpenGL buffer object for writing from CUDA
    float *dptr;
    checkCudaErrors(hipGraphicsMapResources(1, vbo_resource, 0));
    size_t num_bytes;
    checkCudaErrors(hipGraphicsResourceGetMappedPointer((void **)&dptr, &num_bytes,
                                                         *vbo_resource));
    //printf("CUDA mapped VBO: May access %ld bytes\n", num_bytes);


    // execute the kernel
    dim3 grid2D(N/1024, 1, 1);
    dim3 block2D(1024, 1, 1);    
    dim3 grid3D(N / 32, N / 32, 1);
    dim3 block3D(32, 32, 1);
    dim3 gridReduce(N * N / 1024, 1, 1);
    dim3 blockReduce(1024, 1, 1);

    size_t shm_size = 1024 * sizeof(float);

    kernel_prepare_move<<<grid3D, block3D>>>(d_x, d_y, d_vx, d_vy, d_tempx, d_tempy, d_count);
    kernel_reduce3D<<<gridReduce, blockReduce, shm_size>>>(d_tempx, d_tempx);
    kernel_reduce3D<<<gridReduce, blockReduce, shm_size>>>(d_tempy, d_tempy);
    kernel_reduce3D<<<gridReduce, blockReduce, shm_size>>>(d_count, d_count);
    // {
    //     float *debug = new float[N * N];
    //     hipMemcpy(debug, d_count, N * N * sizeof(float), hipMemcpyDeviceToHost);
    //     for(int i = 0; i < N * N; i++)
    //     {
    //         std::cout << i << ":" << debug[i] << std::endl;
    //     }
    //     delete[] debug;
    //     exit(1);
    // }
    kernel_update_velocity<<<grid2D, block2D>>>(d_vx, d_vy, d_tempx, d_tempy, d_count);

    kernel_normalize_velocity<<<grid2D, block2D>>>(d_vx, d_vy);
    kernel_move<<<grid2D, block2D>>>(d_x, d_y, d_vx, d_vy);
    kernel_display<<<grid2D, block2D>>>(dptr, d_x, d_y, d_vx, d_vy);

    // unmap buffer object
    checkCudaErrors(hipGraphicsUnmapResources(1, vbo_resource, 0));
}

////////////////////////////////////////////////////////////////////////////////
//! Keyboard events handler
////////////////////////////////////////////////////////////////////////////////
void keyboard(unsigned char key, int /*x*/, int /*y*/)
{
    switch (key)
    {
        case (27) :
            #if defined(__APPLE__) || defined(MACOSX)
                exit(EXIT_SUCCESS);
            #else
                glutDestroyWindow(glutGetWindow());
                return;
            #endif
    }
}

////////////////////////////////////////////////////////////////////////////////
//! Mouse event handlers
////////////////////////////////////////////////////////////////////////////////
void mouse(int button, int state, int x, int y)
{
    if (state == GLUT_DOWN)
    {
        mouse_buttons |= 1<<button;
    }
    else if (state == GLUT_UP)
    {
        mouse_buttons = 0;
    }

    mouse_old_x = x;
    mouse_old_y = y;
}

void motion(int x, int y)
{
    float dx, dy;
    dx = (float)(x - mouse_old_x);
    dy = (float)(y - mouse_old_y);

    if (mouse_buttons & 1)
    {
        rotate_x += dy * 0.2f;
        rotate_y += dx * 0.2f;
    }
    else if (mouse_buttons & 4)
    {
        translate_z += dy * 0.01f;
    }

    mouse_old_x = x;
    mouse_old_y = y;
}

void computeFPS()
{
    frameCount++;
    fpsCount++;

    if (fpsCount == fpsLimit)
    {
        avgFPS = 1.f / (sdkGetAverageTimerValue(&timer) / 1000.f);
        fpsCount = 0;
        fpsLimit = (int)MAX(avgFPS, 1.f);

        sdkResetTimer(&timer);
    }

    char fps[256];
    sprintf(fps, "Fish simulation: %3.1f fps (Max 100Hz)", avgFPS);
    glutSetWindowTitle(fps);
}