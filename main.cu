#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

#include <helper_gl.h>
#include <GL/freeglut.h>

#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>

#include <helper_functions.h>
#include <hip/hip_runtime_api.h>         
#include <hip/hip_vector_types.h>

#define MAX_EPSILON_ERROR 10.0f
#define THRESHOLD          0.30f
#define REFRESH_DELAY     10 //ms

////////////////////////////////////////////////////////////////////////////////
//! Parameters
////////////////////////////////////////////////////////////////////////////////
#define N 1024 * 5
#define FISH_LENGTH 8.0f
#define FISH_WIDTH 3.0f

#define MAX_VELOCITY 1.0f
#define MIN_VELOCITY 0.4f

#define MAX_ACCELERATION 0.1f

#define SIGHT_RANGE 225.0f //squared
#define PROTECTED_RANGE 64.0f // squared

#define TURN_FACTOR 1.04f
#define COHESION_FACTOR 0.1f
#define ALIGNMENT_FACTOR 0.1f
#define SEPARATION_FACTOR 0.1f
////////////////////////////////////////////////////////////////////////////////
//! Parameters
////////////////////////////////////////////////////////////////////////////////


#define mapRange(a1,a2,b1,b2,s) (b1 + (s-a1)*(b2-b1)/(a2-a1))

// constants
const int window_width  = 1024;
const int window_height = 960;

const int sea_width    = window_width / 2;
const int sea_height   = window_height / 2;

float *d_x, *d_y, *d_vx, *d_vy, *d_cohesionx, *d_cohesiony, *d_alignmentx, *d_alignmenty, *d_separationx, *d_separationy, *d_count;

// vbo variables
GLuint vbo;
struct hipGraphicsResource *cuda_vbo_resource;
void *d_vbo_buffer = NULL;

int mouse_old_x, mouse_old_y;
int mouse_buttons = 0;
float rotate_x = 0.0, rotate_y = 0.0;
float translate_z = -1.0;

int fpsCount = 0;        // FPS count for averaging
int fpsLimit = 1;        // FPS limit for sampling
int g_Index = 0;
float avgFPS = 0.0f;
unsigned int frameCount = 0;

StopWatchInterface *timer = NULL;

// declarations
bool initGL(int *argc, char **argv);
void initCUDA();
void createVBO(GLuint *vbo, struct hipGraphicsResource **vbo_res,
               unsigned int vbo_res_flags);
void deleteVBO(GLuint *vbo, struct hipGraphicsResource *vbo_res);

void cleanup();
void keyboard(unsigned char key, int x, int y);
void mouse(int button, int state, int x, int y);
void motion(int x, int y);

void computeFPS();
void timerEvent(int value);
void display();
void runCuda(struct hipGraphicsResource **vbo_resource);

__global__ void kernel_normalize_velocity(float *vx, float *vy)
{
    unsigned int tid = threadIdx.x + blockIdx.x * blockDim.x;
    
    float tvx = vx[tid];
    float tvy = vy[tid];
    
    float speed = sqrt(tvx * tvx + tvy * tvy);
    if(speed > MAX_VELOCITY)
    {
        vx[tid] = MAX_VELOCITY * tvx / speed;
        vy[tid] = MAX_VELOCITY * tvy / speed;
    }
    else if(speed < MIN_VELOCITY)
    {
        vx[tid] = MIN_VELOCITY * tvx / speed;
        vy[tid] = MIN_VELOCITY * tvy / speed;
    }
}

__global__ void kernel_update_velocity(float *x, float *y, float *vx, float *vy, float *cohX, float *cohY, float *sepX, float *sepY, float *alignX, float *alignY, float *count)
{
    unsigned int tid = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int ntid = N * tid;

    if(count[ntid] != 0)
    {
        float tvx = vx[tid];
        float tvy = vy[tid];

        int cnt = count[ntid];

        float nvx = sepX[ntid] * SEPARATION_FACTOR + (alignX[ntid] / cnt - tvx) * ALIGNMENT_FACTOR  + (cohX[ntid] / cnt - x[tid]) * COHESION_FACTOR;
        float nvy = sepY[ntid] * SEPARATION_FACTOR + (alignY[ntid] / cnt - tvy) * ALIGNMENT_FACTOR  + (cohY[ntid] / cnt - y[tid]) * COHESION_FACTOR;

        float d = sqrt(nvx * nvx + nvy * nvy);


        vx[tid] = tvx + MAX_ACCELERATION / d * (nvx - tvx);
        vy[tid] = tvy + MAX_ACCELERATION / d * (nvy - tvy);


        // vx[tid] = nvx;
        // vy[tid] = nvy;
    }
}

__global__ void reduce(float *data)
{
    extern __shared__ float sdata[];
    // each thread loads one element from global to shared mem
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;
    sdata[tid] = data[i];
    __syncthreads();
    // do reduction in shared mem
    for (unsigned int s=blockDim.x/2; s>0; s>>=1) {
        if (tid < s) {
            sdata[tid] += sdata[tid + s];
        }
        __syncthreads();
    }
    // write result for this block to global mem
    if (tid == 0) data[blockDim.x * blockIdx.x] = sdata[0];
}

__global__ void finish_reduce(float *data, int count)
{
    unsigned int tid = threadIdx.x + blockDim.x * blockIdx.x;
    int sum = 0;
    for(int i = 0; i < count; i++)
    {
        sum += data[tid * N + i * 1024];
    }
    data[tid * N] = sum;
}

__global__ void kernel_prepare_move(float *x, float *y, float *vx, float *vy, float *cohX, float *cohY, float *sepX, float *sepY, float *alignX, float *alignY, float *count)
{
    unsigned int tidx = threadIdx.x + blockIdx.x * blockDim.x; // Fish number tidx is ANOTHER fish
    unsigned int tidy = threadIdx.y + blockIdx.y * blockDim.y; // Fish number tidy is MY fish

    float dx = x[tidy] - x[tidx];
    float dy = y[tidy] - y[tidx];

    float d = dx * dx + dy * dy;

    unsigned int index = tidy * N + tidx;

    if(d < SIGHT_RANGE && tidx != tidy)
    {
        cohX[index] = x[tidx];
        cohY[index] = y[tidx];
        alignX[index] = vx[tidx];
        alignY[index] = vy[tidx];

        count[index] = 1;
        if(d < PROTECTED_RANGE)
        {
            float dsqrt = sqrt(d);

            sepX[index] = dx / dsqrt;
            sepY[index] = dy / dsqrt;
        }
        else
        {
            sepX[index] = 0;
            sepY[index] = 0;
        }
    }
    else
    {
        cohX[index] = 0;
        cohY[index] = 0;        
        sepX[index] = 0;
        sepY[index] = 0;
        alignX[index] = 0;
        alignY[index] = 0;

        count[index] = 0;
    }

}

__global__ void kernel_move(float *x, float *y, float *vx, float *vy)
{
    unsigned int tid = threadIdx.x + blockIdx.x * blockDim.x;
    float nx = x[tid] + vx[tid];
    float ny = y[tid] + vy[tid];

    // repair X velocity
    if(nx < -sea_width)
        vx[tid] = vx[tid] + TURN_FACTOR;
    else if(nx > sea_width)
        vx[tid] = vx[tid] - TURN_FACTOR;
    // repair Y velocity
    if(ny < -sea_height)
        vy[tid] = vy[tid] + TURN_FACTOR;
    else if(ny > sea_height)
        vy[tid] = vy[tid] - TURN_FACTOR;
    x[tid] = nx;
    y[tid] = ny;
    
}

__global__ void kernel_display(float *pos, float *x, float *y, float *vx, float *vy)
{
    unsigned int tid = threadIdx.x + blockIdx.x * blockDim.x;
    float p1X = x[tid];
    float p1Y = y[tid];

    float tvx = vx[tid];
    float tvy = vy[tid];

    float coef = FISH_LENGTH / sqrt(tvx * tvx + tvy * tvy);
    float tempX = coef * tvx;
    float tempY = coef * tvy;

    float p2X = p1X - tempX - tempY * FISH_WIDTH / FISH_LENGTH;
    float p2Y = p1Y - tempY + tempX * FISH_WIDTH / FISH_LENGTH;
    float p3X = p1X - tempX + tempY * FISH_WIDTH / FISH_LENGTH;
    float p3Y = p1Y - tempY - tempX * FISH_WIDTH / FISH_LENGTH;

    pos[9 * tid] = p1X / window_width;
    pos[9 * tid + 1] = p1Y / window_height;
    pos[9 * tid + 2] = 0.0f;
    pos[9 * tid + 3] = p2X / window_width;
    pos[9 * tid + 4] = p2Y / window_height;
    pos[9 * tid + 5] = 0.0f;    
    pos[9 * tid + 6] = p3X / window_width;
    pos[9 * tid + 7] = p3Y / window_height;
    pos[9 * tid + 8] = 0.0f;

}

int main(int argc, char **argv)
{
    // Create the CUTIL timer
    sdkCreateTimer(&timer);

    initCUDA();
    initGL(&argc, argv);

    createVBO(&vbo, &cuda_vbo_resource, cudaGraphicsMapFlagsWriteDiscard);
    glutMainLoop();

    return 0;
}

void initCUDA()
{
    float* h_x = new float[N];
    float* h_y = new float[N];
    float* h_vx = new float[N];
    float* h_vy = new float[N];

    // srand(time(NULL));
    srand(0);

    for(int i = 0; i < N; ++i)
    {
        h_x[i] = (rand() % window_width) - sea_width;
        h_y[i] = (rand() % window_height) - sea_height;

        h_vx[i] = mapRange(0, 100, -MAX_VELOCITY, MAX_VELOCITY, rand() % 100);
        h_vy[i] = mapRange(0, 100, -MAX_VELOCITY, MAX_VELOCITY, rand() % 100);
        // printf("Line %d has x: %f y: %f vx: %f vy: %f\n", i, h_x[i], h_y[i], h_vx[i], h_vy[i]);
    }

    hipMalloc(&d_x, N * sizeof(float));
    hipMalloc(&d_y, N * sizeof(float));
    hipMalloc(&d_vx, N * sizeof(float));
    hipMalloc(&d_vy, N * sizeof(float));
    hipMalloc(&d_cohesionx, N * N * sizeof(float));
    hipMalloc(&d_cohesiony, N * N * sizeof(float));    
    hipMalloc(&d_separationx, N * N * sizeof(float));
    hipMalloc(&d_separationy, N * N * sizeof(float));
    hipMalloc(&d_alignmentx, N * N * sizeof(float));
    hipMalloc(&d_alignmenty, N * N * sizeof(float));
    hipMalloc(&d_count, N * N * sizeof(float));

    hipMemcpy(d_x, h_x, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_y, h_y, N * sizeof(float), hipMemcpyHostToDevice);    
    hipMemcpy(d_vx, h_vx, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_vy, h_vy, N * sizeof(float), hipMemcpyHostToDevice);
}

bool initGL(int *argc, char **argv)
{
    glutInit(argc, argv);
    glutInitDisplayMode(GLUT_RGBA | GLUT_DOUBLE);
    glutInitWindowSize(window_width, window_height);
    glutCreateWindow("Fish simulation");
    glutDisplayFunc(display);
    glutKeyboardFunc(keyboard);
    glutMotionFunc(motion);
    glutTimerFunc(REFRESH_DELAY, timerEvent,0);

    // initialize necessary OpenGL extensions
    if (! isGLVersionSupported(2,0))
    {
        fprintf(stderr, "ERROR: Support for necessary OpenGL extensions missing.");
        fflush(stderr);
        return false;
    }

    // default initialization
    glClearColor(0.0, 0.0, 0.0, 1.0);
    glDisable(GL_DEPTH_TEST);

    // viewport
    glViewport(0, 0, window_width, window_height);

    // projection
    glMatrixMode(GL_PROJECTION);
    glLoadIdentity();
    gluPerspective(60.0, (GLfloat)window_width / (GLfloat) window_height, 0.1, 10.0);

    SDK_CHECK_ERROR_GL();

    glutMouseFunc(mouse);
    glutCloseFunc(cleanup);


    return true;
}

void timerEvent(int value)
{
    if (glutGetWindow())
    {
        glutPostRedisplay();
        glutTimerFunc(REFRESH_DELAY, timerEvent,0);
    }
}

void cleanup()
{

    if (vbo)
    {
        deleteVBO(&vbo, cuda_vbo_resource);
    }
    hipFree(d_x);
    hipFree(d_y);
    hipFree(d_vx);
    hipFree(d_vy);
    hipFree(d_separationx);
    hipFree(d_separationy);
    hipFree(d_cohesionx);
    hipFree(d_cohesiony);
    hipFree(d_alignmentx);
    hipFree(d_alignmenty);
    hipFree(d_count);
}

void createVBO(GLuint *vbo, struct hipGraphicsResource **vbo_res,
               unsigned int vbo_res_flags)
{
    assert(vbo);

    // create buffer object
    glGenBuffers(1, vbo);
    glBindBuffer(GL_ARRAY_BUFFER, *vbo);

    // initialize buffer object
    unsigned int size = 9 * N * sizeof(float);
    glBufferData(GL_ARRAY_BUFFER, size, 0, GL_DYNAMIC_DRAW);

    glBindBuffer(GL_ARRAY_BUFFER, 0);

    // register this buffer object with CUDA
    checkCudaErrors(hipGraphicsGLRegisterBuffer(vbo_res, *vbo, vbo_res_flags));

    SDK_CHECK_ERROR_GL();
}

void deleteVBO(GLuint *vbo, struct hipGraphicsResource *vbo_res)
{

    // unregister this buffer object with CUDA
    checkCudaErrors(hipGraphicsUnregisterResource(vbo_res));

    glBindBuffer(1, *vbo);
    glDeleteBuffers(1, vbo);

    *vbo = 0;
}

void display()
{
    sdkStartTimer(&timer);

    // run CUDA kernel to generate vertex positions
    runCuda(&cuda_vbo_resource);

    glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);

    // set view matrix
    glMatrixMode(GL_MODELVIEW);
    glLoadIdentity();
    glTranslatef(0.0, 0.0, translate_z);
    glRotatef(rotate_x, 1.0, 0.0, 0.0);
    glRotatef(rotate_y, 0.0, 1.0, 0.0);

    // render from the vbo
    glBindBuffer(GL_ARRAY_BUFFER, vbo);
    glVertexPointer(3, GL_FLOAT, 0, 0);

    glEnableClientState(GL_VERTEX_ARRAY);
    glColor3f(1.0, 0.0, 0.0);
    glDrawArrays(GL_TRIANGLES, 0, 9 * N);
    glDisableClientState(GL_VERTEX_ARRAY);

    glutSwapBuffers();
    sdkStopTimer(&timer);
    computeFPS();
}

void runCuda(struct hipGraphicsResource **vbo_resource)
{
    // map OpenGL buffer object for writing from CUDA
    float *dptr;
    checkCudaErrors(hipGraphicsMapResources(1, vbo_resource, 0));
    size_t num_bytes;
    checkCudaErrors(hipGraphicsResourceGetMappedPointer((void **)&dptr, &num_bytes,
                                                         *vbo_resource));
    //printf("CUDA mapped VBO: May access %ld bytes\n", num_bytes);


    // execute the kernel
    dim3 grid2D(N/1024, 1, 1);
    dim3 block2D(1024, 1, 1);    
    dim3 grid3D(N / 32, N / 32, 1);
    dim3 block3D(32, 32, 1);
    dim3 gridReduce(N * N / 1024, 1, 1);
    dim3 blockReduce(1024, 1, 1);

    size_t shm_size = 1024 * sizeof(float);

    kernel_prepare_move<<<grid3D, block3D>>>(d_x, d_y, d_vx, d_vy, d_cohesionx, d_cohesiony, d_separationx, d_separationy, d_alignmentx, d_alignmenty, d_count);
    //     {
    //     float *debug = new float[N * N];
    //     hipMemcpy(debug, d_alignmentx, N * N * sizeof(float), hipMemcpyDeviceToHost);
    //     for(int i = 0; i < N; i++)
    //     {
    //         std::cout << "start";
    //         for(int j = 0; j < N; j++)
    //         {
    //             std::cout << ";" << debug[i * N + j];
    //         }
    //         std::cout << std::endl;
    //     }
    //     delete[] debug;
    // }
    reduce<<<gridReduce, blockReduce, shm_size>>>(d_cohesionx);
    reduce<<<gridReduce, blockReduce, shm_size>>>(d_cohesiony);
    reduce<<<gridReduce, blockReduce, shm_size>>>(d_separationx);
    reduce<<<gridReduce, blockReduce, shm_size>>>(d_separationy);
    reduce<<<gridReduce, blockReduce, shm_size>>>(d_alignmentx);
    reduce<<<gridReduce, blockReduce, shm_size>>>(d_alignmenty);
    reduce<<<gridReduce, blockReduce, shm_size>>>(d_count);
    if(N / 1024 > 1)
    {
        finish_reduce<<<grid2D, block2D>>>(d_cohesionx, N / 1024);
        finish_reduce<<<grid2D, block2D>>>(d_separationx, N / 1024);
        finish_reduce<<<grid2D, block2D>>>(d_alignmentx, N / 1024);
        finish_reduce<<<grid2D, block2D>>>(d_cohesiony, N / 1024);
        finish_reduce<<<grid2D, block2D>>>(d_separationy, N / 1024);
        finish_reduce<<<grid2D, block2D>>>(d_alignmenty, N / 1024);
        finish_reduce<<<grid2D, block2D>>>(d_count, N / 1024);
    }
    // {
    //     float *debug = new float[N * N];
    //     float *debugX = new float[N * N];
    //     float *debugY = new float[N * N];
    //     hipMemcpy(debug, d_count, N * N * sizeof(float), hipMemcpyDeviceToHost);
    //     hipMemcpy(debugX, d_separationx, N * N * sizeof(float), hipMemcpyDeviceToHost);
    //     hipMemcpy(debugY, d_separationx, N * N * sizeof(float), hipMemcpyDeviceToHost);
    //     for(int i = 0; i < N; i++)
    //     {
    //         // for(int j = 0; j < N; j++)
    //         // {
    //         //     std::cout << "1)" << i << ":" << j << ";" << debug[i * N + j] << std::endl;
    //         // }
    //         std::cout << i << ";" << debug[i * N] << ";" << debugX[i * N] << ";" << debugY[i * N] << std::endl;
    //     }
    //     delete[] debug;
    //     delete[] debugX;
    //     delete[] debugY;
    //     exit(1);
    // }

    // {
    //     float *debugX = new float[N];
    //     float *debugY = new float[N];
    //     hipMemcpy(debugX, d_vx, N * sizeof(float), hipMemcpyDeviceToHost);
    //     hipMemcpy(debugY, d_vy, N * sizeof(float), hipMemcpyDeviceToHost);
    //     for(int i = 0; i < N; i++)
    //     {
    //         std::cout << "0|" << i << ";" << debugX[i] << ";" << debugY[i] << std::endl;
    //     }
    //     delete[] debugX;
    //     delete[] debugY;
    // }
    kernel_update_velocity<<<grid2D, block2D>>>(d_x, d_y, d_vx, d_vy, d_cohesionx, d_cohesiony, d_separationx, d_separationy, d_alignmentx, d_alignmenty, d_count);
    // {
    //     float *debugX = new float[N];
    //     float *debugY = new float[N];
    //     hipMemcpy(debugX, d_vx, N * sizeof(float), hipMemcpyDeviceToHost);
    //     hipMemcpy(debugY, d_vy, N * sizeof(float), hipMemcpyDeviceToHost);
    //     for(int i = 0; i < N; i++)
    //     {
    //         std::cout << "1|" << i << ";" << debugX[i] << ";" << debugY[i] << std::endl;
    //     }
    //     delete[] debugX;
    //     delete[] debugY;
    // }
    kernel_normalize_velocity<<<grid2D, block2D>>>(d_vx, d_vy);
    // {
    //     float *debugX = new float[N];
    //     float *debugY = new float[N];
    //     hipMemcpy(debugX, d_vx, N * sizeof(float), hipMemcpyDeviceToHost);
    //     hipMemcpy(debugY, d_vy, N * sizeof(float), hipMemcpyDeviceToHost);
    //     for(int i = 0; i < N; i++)
    //     {
    //         std::cout << "2|" << i << ";" << debugX[i] << ";" << debugY[i] << std::endl;
    //     }
    //     delete[] debugX;
    //     delete[] debugY;
    //     exit(1);
    // }
    kernel_move<<<grid2D, block2D>>>(d_x, d_y, d_vx, d_vy);
    kernel_display<<<grid2D, block2D>>>(dptr, d_x, d_y, d_vx, d_vy);

    // unmap buffer object
    checkCudaErrors(hipGraphicsUnmapResources(1, vbo_resource, 0));
}

////////////////////////////////////////////////////////////////////////////////
//! Keyboard events handler
////////////////////////////////////////////////////////////////////////////////
void keyboard(unsigned char key, int /*x*/, int /*y*/)
{
    switch (key)
    {
        case (27) :
            #if defined(__APPLE__) || defined(MACOSX)
                exit(EXIT_SUCCESS);
            #else
                glutDestroyWindow(glutGetWindow());
                return;
            #endif
    }
}

////////////////////////////////////////////////////////////////////////////////
//! Mouse event handlers
////////////////////////////////////////////////////////////////////////////////
void mouse(int button, int state, int x, int y)
{
    if (state == GLUT_DOWN)
    {
        mouse_buttons |= 1<<button;
    }
    else if (state == GLUT_UP)
    {
        mouse_buttons = 0;
    }

    mouse_old_x = x;
    mouse_old_y = y;
}

void motion(int x, int y)
{
    float dx, dy;
    dx = (float)(x - mouse_old_x);
    dy = (float)(y - mouse_old_y);

    if (mouse_buttons & 1)
    {
        rotate_x += dy * 0.2f;
        rotate_y += dx * 0.2f;
    }
    else if (mouse_buttons & 4)
    {
        translate_z += dy * 0.01f;
    }

    mouse_old_x = x;
    mouse_old_y = y;
}

void computeFPS()
{
    frameCount++;
    fpsCount++;

    if (fpsCount == fpsLimit)
    {
        avgFPS = 1.f / (sdkGetAverageTimerValue(&timer) / 1000.f);
        fpsCount = 0;
        fpsLimit = (int)MAX(avgFPS, 1.f);

        sdkResetTimer(&timer);
    }

    char fps[256];
    sprintf(fps, "Fish simulation: %3.1f fps (Max 100Hz)", avgFPS);
    glutSetWindowTitle(fps);
}